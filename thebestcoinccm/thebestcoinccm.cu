extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "lyra2_params.h"


static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static  uint64_t *d_hash2[MAX_GPUS];

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void Keccak256_cpu_hash_32(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void skeinCube256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);


extern void lyra2v2_cpu_init_VAR_32_32(int thr_id, uint32_t threads,uint64_t *hash);
extern void lyra2v2_cpu_hash_32_VAR_32_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, uint32_t tpb);

extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t target);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash);

extern "C" int LYRA2(void *K, uint64_t kLen, const void *pwd, uint64_t pwdlen, const void *salt, uint64_t saltlen, uint64_t timeCost, uint64_t nRows, uint64_t nCols);

/**
 * CPU version of TheBestCoin algorithm
 */
extern "C" void thebestcoinccm_hash(void *state, const void *input)
{
	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);


	LYRA2(hashB, 32, hashA, 32, hashA, 32, LYRA2_TCOST, LYRA2_ROWS, LYRA2_COLS);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);


	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_thebestcoinccm(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = 256 * 256 * 8;
	uint32_t tpb = 8;
//	bool mergeblakekeccak = false;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);

	// calculate intensity depending of algo params, may not work for some params and GPUs
	unsigned int vram = 1024 * 1024 * 1536; // 1,5GB VRAM
	unsigned int msize = BLOCK_LEN_BYTES * LYRA2_COLS * LYRA2_ROWS; // Matrix size
	intensity = (vram / msize);

	// Values of tpb and intensity can be changed for specific video card to tune performance
	//if (strstr(props.name, "980 Ti"))
	//{
	//	tpb = 10;
	//	intensity = 256 * 256 * 18;
	//}

//    intensity = 1024 * 1024;
//    tpb = 13;
//    intensity = 128 * 1024;
//    tpb = 29;
    intensity = 16 * 1024;
    tpb = 8;

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		//keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		applog(LOG_INFO, "GPU #%d: lyra2 params set to R = %d, C = %d, T = %d", thr_id, LYRA2_ROWS, LYRA2_COLS, LYRA2_TCOST);
		applog(LOG_INFO, "GPU #%d: allocating %d MB memory for DMatrix (%d B per thread)", thr_id, ROW_LEN_BYTES * LYRA2_ROWS * throughput / 1048576, ROW_LEN_BYTES * LYRA2_ROWS);
		applog(LOG_INFO, "GPU #%d: throughput is set to %d, tpb = %d", thr_id, throughput, tpb);

		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], ROW_LEN_BYTES * LYRA2_ROWS * throughput)); // todo is d_hash2 used now?
		lyra2v2_cpu_init_VAR_32_32(thr_id, throughput, d_hash2[thr_id]);
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput));
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);

	do {
		uint32_t foundNonce[2] = { 0, 0 };

		// Separate calculation may be faster on some GPUs
//		if (mergeblakekeccak)
//		{
			blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);

/*		}
		else
		{
			blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
			keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		}
*/

		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);

		lyra2v2_cpu_hash_32_VAR_32_32(thr_id, throughput, pdata[19], d_hash[thr_id], tpb);

		hipError_t cer = hipGetLastError();
		if (cer != hipSuccess) {
			applog(LOG_INFO, "Cuda error (thebestcoinccm_cpu_hash_32): %s", hipGetErrorString(cer));
			scan_abort_flag = true;
			break;
		}

		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce, ptarget[7]);

		if (foundNonce[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			thebestcoinccm_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0xffffffff)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
